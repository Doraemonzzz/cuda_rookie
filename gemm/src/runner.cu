#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "runner.cuh"
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iomanip>

float get_sec() {
  struct timeval time;
  gettimeofday(&time, NULL);
  return (1e6 * time.tv_sec + time.tv_usec);
}

float cpu_elapsed_time(float &beg, float &end) { return 1.0e-6 * (end - beg); }

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

void CudaDeviceInfo() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

void randomize_matrix(float *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  srand(time.tv_usec);
  for (int i = 0; i < N; i++) {
    float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

void range_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = i;
  }
}

void zero_init_matrix(float *mat, int N) {
  for (int i = 0; i < N; i++) {
    mat[i] = 0.0;
  }
}

void copy_matrix(const float *src, float *dest, int N) {
  int i;
  for (i = 0; src + i && dest + i && i < N; i++)
    *(dest + i) = *(src + i);
  if (i != N)
    printf("copy failed at %d while there are %d elements in total.\n", i, N);
}

void print_matrix(const float *A, int M, int N, std::ofstream &fs) {
  int i;
  fs << std::setprecision(2)
     << std::fixed; // Set floating-point precision and fixed notation
  fs << "[";
  for (i = 0; i < M * N; i++) {
    if ((i + 1) % N == 0)
      fs << std::setw(5) << A[i]; // Set field width and write the value
    else
      fs << std::setw(5) << A[i] << ", ";
    if ((i + 1) % N == 0) {
      if (i + 1 < M * N)
        fs << ";\n";
    }
  }
  fs << "]\n";
}

bool verify_matrix(float *matRef, float *matOut, int N) {
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) {
    diff = std::fabs(matRef[i] - matOut[i]);
    if (diff > 0.01) {
      printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
             matRef[i], matOut[i], diff, i);
      return false;
    }
  }
  return true;
}

int div_ceil(int numerator, int denominator) {
  std::div_t res = std::div(numerator, denominator);
  return res.rem ? (res.quot + 1) : res.quot;
}

void runCublasFP32(hipblasHandle_t handle, int M, int N, int K, float alpha,
                   float *A, float *B, float beta, float *C) {
  // cuBLAS uses column-major order. So we change the order of our row-major A &
  // B, since (B^T*A^T)^T = (A*B)
  // This runs cuBLAS in full fp32 mode
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void runCublasBF16(hipblasHandle_t handle, int M, int N, int K, float alpha,
                   float *A, float *B, float beta, float *C) {
  // This runs cuBLAS with mixed precision (performing the mul with operands
  // downcast to bf16), which is ~4x faster
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N,
               HIPBLAS_COMPUTE_32F_FAST_16BF, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void runCublasTF32(hipblasHandle_t handle, int M, int N, int K, float alpha,
                   float *A, float *B, float beta, float *C) {
  // This runs cuBLAS with mixed precision (performing the mul with operands
  // downcast to bf16), which is ~4x faster
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N,
               HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void run_sgemm_naive(int M, int N, int K, float alpha, float *A, float *B,
                     float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32, 32);
  sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_coalesce(int M, int N, int K, float alpha, float *A, float *B,
                        float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  sgemm_global_mem_coalesce<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_shared_mem_block(int M, int N, int K, float alpha, float *A,
                                float *B, float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  // L1 cache becomes useless, since we access GMEM only via SMEM, so we carve
  // out all of L1 to SMEM. This doesn't currently make a difference, since
  // occupancy is limited by reg and thread count, but it's good to do anyway.
  hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shared_mem_block<32>),
                       hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);
  sgemm_shared_mem_block<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void runSgemm1DBlocktiling(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  const uint BM = 64;
  const uint BN = 64;
  const uint BK = 8;
  const uint TM = 8;
  dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
  dim3 blockDim((BM * BN) / TM);
  sgemm1DBlocktiling<BM, BN, BK, TM>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void runSgemm2DBlocktiling(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  const uint BK = 8;
  const uint TM = 8;
  const uint TN = 8;
  if (M >= 128 and N >= 128) {
    const uint BM = 128;
    const uint BN = 128;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemm2DBlocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  } else {
    // this is a hacky solution to the underlying problem
    // of not having proper bounds checking in the kernel
    const uint BM = 64;
    const uint BN = 64;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemm2DBlocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  }
}

void runSgemmVectorize(int M, int N, int K, float alpha, float *A, float *B,
                       float beta, float *C) {
  const uint BK = 8;
  const uint TM = 8;
  const uint TN = 8;
  if (M >= 128 and N >= 128) {
    const uint BM = 128;
    const uint BN = 128;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmVectorize<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  } else {
    // this is a hacky solution to the underlying problem
    // of not having proper bounds checking in the kernel
    const uint BM = 64;
    const uint BN = 64;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmVectorize<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  }
}

void runSgemmResolveBankConflicts(int M, int N, int K, float alpha, float *A,
                                  float *B, float beta, float *C) {
  const uint BK = 8;
  const uint TM = 8;
  const uint TN = 8;
  if (M >= 128 and N >= 128) {
    const uint BM = 128;
    const uint BN = 128;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmResolveBankConflicts<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  } else {
    // this is a hacky solution to the underlying problem
    // of not having proper bounds checking in the kernel
    const uint BM = 64;
    const uint BN = 64;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmResolveBankConflicts<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  }
}

void runSgemmResolveBankExtraCol(int M, int N, int K, float alpha, float *A,
                                 float *B, float beta, float *C) {
  const uint BK = 8;
  const uint TM = 8;
  const uint TN = 8;
  if (M >= 128 and N >= 128) {
    const uint BM = 128;
    const uint BN = 128;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmResolveBankExtraCol<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  } else {
    // this is a hacky solution to the underlying problem
    // of not having proper bounds checking in the kernel
    const uint BM = 64;
    const uint BN = 64;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemmResolveBankExtraCol<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  }
}

void runSgemmAutotuned(int M, int N, int K, float alpha, float *A, float *B,
                       float beta, float *C) {
  // A100
  // const uint K9_BK = 16;
  // const uint K9_TM = 4;
  // const uint K9_TN = 4;
  // const uint K9_BM = 64;
  // const uint K9_BN = 64;
  // A6000
  const uint K9_BK = 16;
  const uint K9_TM = 8;
  const uint K9_TN = 8;
  const uint K9_BM = 128;
  const uint K9_BN = 128;
  dim3 blockDim(K9_NUM_THREADS);

  static_assert(
      (K9_NUM_THREADS * 4) % K9_BK == 0,
      "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization issues "
      "during GMEM->SMEM tiling (loading only parts of the final row of Bs "
      "during each iteraion)");
  static_assert(
      (K9_NUM_THREADS * 4) % K9_BN == 0,
      "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization issues "
      "during GMEM->SMEM tiling (loading only parts of the final row of As "
      "during each iteration)");
  static_assert(
      K9_BN % (16 * K9_TN) == 0,
      "K9_BN must be a multiple of 16*K9_TN to avoid quantization effects");
  static_assert(
      K9_BM % (16 * K9_TM) == 0,
      "K9_BM must be a multiple of 16*K9_TM to avoid quantization effects");
  static_assert((K9_BM * K9_BK) % (4 * K9_NUM_THREADS) == 0,
                "K9_BM*K9_BK must be a multiple of 4*256 to vectorize loads");
  static_assert((K9_BN * K9_BK) % (4 * K9_NUM_THREADS) == 0,
                "K9_BN*K9_BK must be a multiple of 4*256 to vectorize loads");

  dim3 gridDim(CEIL_DIV(N, K9_BN), CEIL_DIV(M, K9_BM));
  sgemmAutotuned<K9_BM, K9_BN, K9_BK, K9_TM, K9_TN>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void runSgemmWarptiling(int M, int N, int K, float alpha, float *A, float *B,
                        float beta, float *C) {
  // Settings for A100
  // const uint K10_NUM_THREADS = 128;
  // const uint K10_BN = 128;
  // const uint K10_BM = 64;
  // const uint K10_BK = 16;
  // const uint K10_WN = 64;
  // const uint K10_WM = 32;
  // const uint K10_WNITER = 1;
  // const uint K10_TN = 4;
  // const uint K10_TM = 4;
  // Settings for A6000
  const uint K10_NUM_THREADS = 128;
  const uint K10_BN = 128;
  const uint K10_BM = 128;
  const uint K10_BK = 16;
  const uint K10_WN = 64;
  const uint K10_WM = 64;
  const uint K10_WNITER = 4;
  const uint K10_TN = 4;
  const uint K10_TM = 8;
  dim3 blockDim(K10_NUM_THREADS);

  constexpr uint NUM_WARPS = K10_NUM_THREADS / 32;

  // warptile in threadblocktile
  static_assert((K10_BN % K10_WN == 0) and (K10_BM % K10_WM == 0));
  static_assert((K10_BN / K10_WN) * (K10_BM / K10_WM) == NUM_WARPS);

  // threads in warpsubtile
  static_assert((K10_WM * K10_WN) % (WARPSIZE * K10_TM * K10_TN * K10_WNITER) ==
                0);
  constexpr uint K10_WMITER =
      (K10_WM * K10_WN) / (32 * K10_TM * K10_TN * K10_WNITER);
  // warpsubtile in warptile
  static_assert((K10_WM % K10_WMITER == 0) and (K10_WN % K10_WNITER == 0));

  static_assert((K10_NUM_THREADS * 4) % K10_BK == 0,
                "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of Bs during each iteraion)");
  static_assert((K10_NUM_THREADS * 4) % K10_BN == 0,
                "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of As during each iteration)");
  static_assert(K10_BN % (16 * K10_TN) == 0,
                "BN must be a multiple of 16*TN to avoid quantization effects");
  static_assert(K10_BM % (16 * K10_TM) == 0,
                "BM must be a multiple of 16*TM to avoid quantization effects");
  static_assert((K10_BM * K10_BK) % (4 * K10_NUM_THREADS) == 0,
                "BM*BK must be a multiple of 4*256 to vectorize loads");
  static_assert((K10_BN * K10_BK) % (4 * K10_NUM_THREADS) == 0,
                "BN*BK must be a multiple of 4*256 to vectorize loads");

  dim3 gridDim(CEIL_DIV(N, K10_BN), CEIL_DIV(M, K10_BM));
  sgemmWarptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM,
                  K10_TN, K10_NUM_THREADS>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void runSgemmDoubleBuffering(int M, int N, int K, float alpha, float *A,
                             float *B, float beta, float *C) {
  // Settings for A100
  // const uint K11_NUM_THREADS = 256;
  // const uint K11_BN = 128;
  // const uint K11_BM = 64;
  // const uint K11_BK = 16;
  // const uint K11_WN = 32;
  // const uint K11_WM = 32;
  // const uint K11_WNITER = 2;
  // const uint K11_TN = 4;
  // const uint K11_TM = 4;
  // Settings for A6000
  const uint K11_NUM_THREADS = 256;
  const uint K11_BN = 256;
  const uint K11_BM = 128;
  const uint K11_BK = 16;
  const uint K11_WN = 32;
  const uint K11_WM = 128;
  const uint K11_WNITER = 1;
  const uint K11_TN = 8;
  const uint K11_TM = 8;
  dim3 blockDim(K11_NUM_THREADS);

  constexpr uint NUM_WARPS = K11_NUM_THREADS / 32;

  // warptile in threadblocktile
  static_assert((K11_BN % K11_WN == 0) and (K11_BM % K11_WM == 0));
  static_assert((K11_BN / K11_WN) * (K11_BM / K11_WM) == NUM_WARPS);

  // threads in warpsubtile
  static_assert((K11_WM * K11_WN) % (WARPSIZE * K11_TM * K11_TN * K11_WNITER) ==
                0);
  constexpr uint K11_WMITER =
      (K11_WM * K11_WN) / (32 * K11_TM * K11_TN * K11_WNITER);
  // warpsubtile in warptile
  static_assert((K11_WM % K11_WMITER == 0) and (K11_WN % K11_WNITER == 0));

  static_assert((K11_NUM_THREADS / 2 * 4) % K11_BK == 0,
                "NUM_THREADS*4 must be multiple of BK to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of Bs during each iteraion)");
  static_assert((K11_NUM_THREADS / 2 * 4) % K11_BN == 0,
                "NUM_THREADS*4 must be multiple of BN to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of As during each iteration)");
  static_assert(K11_BN % (16 * K11_TN) == 0,
                "BN must be a multiple of 16*TN to avoid quantization effects");
  static_assert(K11_BM % (16 * K11_TM) == 0,
                "BM must be a multiple of 16*TM to avoid quantization effects");
  static_assert((K11_BM * K11_BK) % (4 * K11_NUM_THREADS / 2) == 0,
                "BM*BK must be a multiple of 4*256 to vectorize loads");
  static_assert((K11_BN * K11_BK) % (4 * K11_NUM_THREADS / 2) == 0,
                "BN*BK must be a multiple of 4*256 to vectorize loads");

  dim3 gridDim(CEIL_DIV(N, K11_BN), CEIL_DIV(M, K11_BM));
  sgemmDoubleBuffering<K11_BM, K11_BN, K11_BK, K11_WM, K11_WN, K11_WNITER,
                       K11_TM, K11_TN, K11_NUM_THREADS>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void runSgemmDoubleBuffering2(int M, int N, int K, float alpha, float *A,
                              float *B, float beta, float *C) {
  // Settings for A6000
  const uint K12_NUM_THREADS = 128;
  const uint K12_BN = 128;
  const uint K12_BM = 128;
  const uint K12_BK = 16;
  const uint K12_WN = 64;
  const uint K12_WM = 64;
  const uint K12_WNITER = 4;
  const uint K12_TN = 4;
  const uint K12_TM = 8;
  dim3 blockDim(K12_NUM_THREADS);

  constexpr uint NUM_WARPS = K12_NUM_THREADS / 32;

  // warptile in threadblocktile
  static_assert((K12_BN % K12_WN == 0) and (K12_BM % K12_WM == 0));
  static_assert((K12_BN / K12_WN) * (K12_BM / K12_WM) == NUM_WARPS);

  // threads in warpsubtile
  static_assert((K12_WM * K12_WN) % (WARPSIZE * K12_TM * K12_TN * K12_WNITER) ==
                0);
  constexpr uint K12_WMITER =
      (K12_WM * K12_WN) / (32 * K12_TM * K12_TN * K12_WNITER);
  // warpsubtile in warptile
  static_assert((K12_WM % K12_WMITER == 0) and (K12_WN % K12_WNITER == 0));

  static_assert((K12_NUM_THREADS * 4) % K12_BK == 0,
                "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of Bs during each iteraion)");
  static_assert((K12_NUM_THREADS * 4) % K12_BN == 0,
                "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization "
                "issues during GMEM->SMEM tiling (loading only parts of the "
                "final row of As during each iteration)");
  static_assert(K12_BN % (16 * K12_TN) == 0,
                "BN must be a multiple of 16*TN to avoid quantization effects");
  static_assert(K12_BM % (16 * K12_TM) == 0,
                "BM must be a multiple of 16*TM to avoid quantization effects");
  static_assert((K12_BM * K12_BK) % (4 * K12_NUM_THREADS) == 0,
                "BM*BK must be a multiple of 4*256 to vectorize loads");
  static_assert((K12_BN * K12_BK) % (4 * K12_NUM_THREADS) == 0,
                "BN*BK must be a multiple of 4*256 to vectorize loads");

  dim3 gridDim(CEIL_DIV(N, K12_BN), CEIL_DIV(M, K12_BM));
  runSgemmDoubleBuffering2<K12_BM, K12_BN, K12_BK, K12_WM, K12_WN, K12_WNITER,
                           K12_TM, K12_TN, K12_NUM_THREADS>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A,
                float *B, float beta, float *C, hipblasHandle_t handle) {
  switch (kernel_num) {
  case 0:
    runCublasFP32(handle, M, N, K, alpha, A, B, beta, C);
    break;
  case 1:
    run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
    break;
  case 2:
    run_sgemm_coalesce(M, N, K, alpha, A, B, beta, C);
    break;
  case 3:
    run_sgemm_shared_mem_block(M, N, K, alpha, A, B, beta, C);
    break;
  case 4:
    runSgemm1DBlocktiling(M, N, K, alpha, A, B, beta, C);
    break;
  case 5:
    runSgemm2DBlocktiling(M, N, K, alpha, A, B, beta, C);
    break;
  case 6:
    runSgemmVectorize(M, N, K, alpha, A, B, beta, C);
    break;
  case 7:
    runSgemmResolveBankConflicts(M, N, K, alpha, A, B, beta, C);
    break;
  case 8:
    runSgemmResolveBankExtraCol(M, N, K, alpha, A, B, beta, C);
    break;
  case 9:
    runSgemmAutotuned(M, N, K, alpha, A, B, beta, C);
    break;
  case 10:
    runSgemmWarptiling(M, N, K, alpha, A, B, beta, C);
    break;
  case 11:
    runSgemmDoubleBuffering(M, N, K, alpha, A, B, beta, C);
    break;
  case 12:
    runSgemmDoubleBuffering2(M, N, K, alpha, A, B, beta, C);
    break;
  default:
    throw std::invalid_argument("Unknown kernel number");
  }
}